#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/types.h>

#include <ATen/cuda/CUDAApplyUtils.cuh>

template <typename scalar_t>
static __global__ void
fused_bias_act_kernel(scalar_t *out, const scalar_t *p_x, const scalar_t *p_b,
					  const scalar_t *p_ref, scalar_t alpha,
					  scalar_t scale, int loop_x, int size_x, int step_b,
					  int size_b, int use_bias, int use_ref) {
	int xi = blockIdx.x * loop_x * blockDim.x + threadIdx.x;

	for (int loop_idx = 0; loop_idx < loop_x && xi < size_x;
		 loop_idx++, xi += blockDim.x) {
		scalar_t x = p_x[xi];

		if (use_bias) {
			x += p_b[(xi / step_b) % size_b];
		}

		scalar_t y;

		if (use_ref) {
			y = (p_ref[xi] > 0.0) ? x : x * alpha;
		} else {
			y = (x > 0.0) ? x : x * alpha;
		}

		out[xi] = y * scale;
	}
}

torch::Tensor fused_bias_act_op(const torch::Tensor &input,
								const torch::Tensor &bias,
								const torch::Tensor &refer,
								float alpha, float scale) {
	// Get current thread's CUDA device (set by DeviceGuard) and stream
	int curDevice = -1;
	hipGetDevice(&curDevice);
	hipStream_t stream = at::cuda::getCurrentCUDAStream();

	// Seems extra given that input is already verified to be contiguous
	auto x	 = input.contiguous();
	auto b	 = bias.contiguous();
	auto ref = refer.contiguous();

	// Check if bias and ref are provided
	int use_bias = b.numel() ? 1 : 0;
	int use_ref	 = ref.numel() ? 1 : 0;

	int size_x = x.numel();
	int size_b = b.numel();
	int step_b = 1;

	for (int i = 2; i < x.dim(); i++) {
		step_b *= x.size(i);
	}

	int loop_x	   = 4;
	int block_size = 4 * 32;
	int grid_size  = (size_x - 1) / (loop_x * block_size) + 1;

	auto y = torch::empty_like(x);

	AT_DISPATCH_FLOATING_TYPES_AND_HALF(
		x.scalar_type(), "fused_bias_act_kernel", [&] {
			fused_bias_act_kernel<scalar_t><<<grid_size, block_size, 0, stream>>>(
				y.data_ptr<scalar_t>(), x.data_ptr<scalar_t>(),
				b.data_ptr<scalar_t>(), ref.data_ptr<scalar_t>(), alpha,
				scale, loop_x, size_x, step_b, size_b, use_bias, use_ref);
		});

	return y;
}